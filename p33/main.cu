#include "hip/hip_runtime.h"
/* ECE 6122 Final Project - Cuda Part
* Writer: Haoran Li, Yingqiao Zheng
* Tested on Pace, College of Computing at Georgia Tech
*/

#include <iostream>
#include <cmath>
#include <time.h>
#include <string>
#include <chrono>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "complex.h"
#include "input_image.h"

#define TX 8
#define TY 8
#define PI 3.14159265358979
//#define GRAPHDIS
#define FILE_PATH_MAX 1024

using std::cout;
using std::endl;
using std::string;
using std::to_string;

__global__ void DFTComputeRow(float* d_real_output, float* d_imag_output, float* d_real_data, float* d_imag_data, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row * width + col;
    float temp_real = 0, temp_imag = 0;

    if(row >= 0 && row < height && col >= 0 && col < width) {
        int start_idx = row * width;
        for(int k = 0; k < width; k++) {
            float W_real = cos(2 * PI * col * k / width), W_imag = -sin(2 * PI * col * k / width);
            //printf("Index: %d, row: %d, col: %d, W_real: %f, W_imag: %f, real_data: %f, imag_data: %f", index, row, col, W_real, W_imag, d_real_data[start_idx + k], d_imag_data[start_idx + k]);
            temp_real += W_real * d_real_data[start_idx + k] - W_imag * d_imag_data[start_idx + k];
            temp_imag += W_real * d_imag_data[start_idx + k] + W_imag * d_real_data[start_idx + k];
        }
        d_real_output[index] = temp_real;
        d_imag_output[index] = temp_imag;
        __syncthreads();
    }

}


__global__ void DFTComputeCol(float* d_real_output, float* d_imag_output, float* d_real_data, float* d_imag_data, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row * width + col;
    float temp_real = 0, temp_imag = 0;
    //if(index == 0) printf("height: %d\n", height);
    if(row >= 0 && row < height && col >= 0 && col < width) {
        int start_idx = col;
        for(int k = 0; k < height; k++) {
            float W_real = cos(2 * PI * row * k / height), W_imag = -sin(2 * PI * row * k/ height);
            temp_real += W_real * d_real_data[start_idx + k * width] - W_imag * d_imag_data[start_idx + k * width];
            temp_imag += W_real * d_imag_data[start_idx + k * width] + W_imag * d_real_data[start_idx + k * width];
        }
        d_real_output[index] = temp_real;
        d_imag_output[index] = temp_imag;
        //printf("row: %d, col: %d, index: %d, compute: %f, %f \n", row, col, index, d_real_output[index], d_imag_output[index]);
        __syncthreads();
    }

}


__global__ void iDFTComputeRow(float* d_real_output, float* d_imag_output, float* d_real_data, float* d_imag_data, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row * width + col;
    float temp_real = 0, temp_imag = 0;

    if(row >= 0 && row < height && col >= 0 && col < width) {
        int start_idx = row * width;
        for(int k = 0; k < width; k++) {
            float W_real = cos(2 * PI * col * k / width), W_imag = sin(2 * PI * col * k/ width);
            temp_real += W_real * d_real_data[start_idx + k] - W_imag * d_imag_data[start_idx + k];
            temp_imag += W_real * d_imag_data[start_idx + k] + W_imag * d_real_data[start_idx + k];
        }
        temp_real /= width;
        temp_imag /= width;
        d_real_output[index] = temp_real;
        d_imag_output[index] = temp_imag;
        __syncthreads();
    }

}


__global__ void iDFTComputeCol(float* d_real_output, float* d_imag_output, float* d_real_data, float* d_imag_data, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row * width + col;
    float temp_real = 0, temp_imag = 0;
    if(row >= 0 && row < height && col >= 0 && col < width) {
        int start_idx = col;
        for(int k = 0; k < height; k++) {
            float W_real = cos(2 * PI * row * k / height), W_imag = sin(2 * PI * row * k/ height);
            temp_real += W_real * d_real_data[start_idx + k * width] - W_imag * d_imag_data[start_idx + k * width];
            temp_imag += W_real * d_imag_data[start_idx + k * width] + W_imag * d_real_data[start_idx + k * width];
        }
        temp_real /= width;
        temp_imag /= width;
        d_real_output[index] = temp_real;
        d_imag_output[index] = temp_imag;
        __syncthreads();
    }

}


int main(int argc, char** argv) {
    if(argc != 4) {
        cout << "ERROR! Incorrect input parameters format! Please enter parameter like this:" << endl;
        cout << "time ./NAME [forward/reverse] [INPUTFILE] [OUTPUTFILE]" << endl;
        exit(0);
    }

    string execution = argv[1];
    if(execution != "forward" && execution != "reverse") {
        cout << "ERROR! Invalid execution type, you can only specify either 'forward' or 'reverse'!" << endl;
        exit(0);
    }

    auto start = std::chrono::system_clock::now();
    char buf[FILE_PATH_MAX];
    if(getcwd(buf, FILE_PATH_MAX) == 0) {
        cout << "ERROR! Unable to get current working directory!" << endl;
        exit(0);
    }
    string cur_work_path = buf;
    string file_path = cur_work_path + "/" + argv[2];
    string out_file_path = argv[3];
    InputImage image(file_path.c_str());

    int width = image.get_width();
    int height = image.get_height();
    int size = width * height;
    Complex *data;
    float *real_data;
    float *imag_data;
    float *d_real_data;
    float *d_imag_data;
    float *d_real_output;
    float *d_imag_output;

    data = (Complex*)malloc(size * sizeof(Complex));
    real_data = (float*)malloc(size * sizeof(float));
    imag_data = (float*)malloc(size * sizeof(float));

    data = image.get_image_data();

    for(int cnt = 0; cnt < size; cnt++) {
        real_data[cnt] = data[cnt].real;
        imag_data[cnt] = data[cnt].imag;
    }

    const int BX = (width + TX - 1) / TX;
    const int BY = (height + TY - 1) / TY;
    dim3 blocks(BX, BY);
    dim3 threads(TX, TY);

#ifdef GRAPHDIS
    for(int idx = 0; idx < size; idx++) {
        cout << data[idx] << " ";
        if(idx % width == width - 1) {
            cout << endl;
        }
    }
#endif

    hipMalloc((void**)&d_real_data, size * sizeof(float));
    hipMalloc((void**)&d_imag_data, size * sizeof(float));
    hipMalloc((void**)&d_real_output, size * sizeof(float));
    hipMalloc((void**)&d_imag_output, size * sizeof(float));

    hipMemcpy(d_real_data, real_data, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_imag_data, imag_data, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_real_output, real_data, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_imag_output, imag_data, size * sizeof(float), hipMemcpyHostToDevice);

    //DFT
    DFTComputeRow<<<blocks, threads>>>(d_real_output, d_imag_output, d_real_data, d_imag_data, width, height);

    hipMemcpy(real_data, d_real_output, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(imag_data, d_imag_output, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_real_data, real_data, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_imag_data, imag_data, size * sizeof(float), hipMemcpyHostToDevice);
        
    DFTComputeCol<<<blocks, threads>>>(d_real_output, d_imag_output, d_real_data, d_imag_data, width, height);

    hipMemcpy(real_data, d_real_output, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(imag_data, d_imag_output, size * sizeof(float), hipMemcpyDeviceToHost);

    if(execution == "reverse") {   //iDFT
        hipMemcpy(d_real_data, real_data, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_imag_data, imag_data, size * sizeof(float), hipMemcpyHostToDevice);

        iDFTComputeRow<<<blocks, threads>>>(d_real_output, d_imag_output, d_real_data, d_imag_data, width, height);

        hipMemcpy(real_data, d_real_output, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(imag_data, d_imag_output, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(d_real_data, real_data, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_imag_data, imag_data, size * sizeof(float), hipMemcpyHostToDevice);
        
        iDFTComputeCol<<<blocks, threads>>>(d_real_output, d_imag_output, d_real_data, d_imag_data, width, height);

        hipMemcpy(real_data, d_real_output, size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(imag_data, d_imag_output, size * sizeof(float), hipMemcpyDeviceToHost);
    }

    auto end = std::chrono::system_clock::now();

#ifdef GRAPHDIS
    cout << "START PRINT CPU RESULT!!" << endl;
#endif
    for(int cnt = 0; cnt < size; cnt++) {
        data[cnt].real = real_data[cnt];
        data[cnt].imag = imag_data[cnt];
#ifdef GRAPHDIS
        cout << "(" << real_data[cnt] << ", " << imag_data[cnt] << ")" << "  ";
        if(cnt % width == width - 1) {
            cout << endl;
        }
#endif
    }
    std::chrono::duration<double> duration = end - start;
    cout << "System Running Time: " << 1000 * duration.count() << "ms" << endl;
    image.save_image_data(out_file_path.c_str(), data, width, height);

    free(data); free(real_data); free(imag_data);
    hipFree(d_real_data); hipFree(d_imag_data); hipFree(d_real_output); hipFree(d_imag_output);
    return 0;
}
